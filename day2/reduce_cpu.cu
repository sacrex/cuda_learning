#include "common.h"

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 20;
void timing(const real *x, const int N);
real reduce(const real *x, const int N);

int main()
{
    const int N = 100000000;
    real *x = (real *)malloc(sizeof(real) * N);
    for (int n = 0; n < N; ++n) {
        x[n] = 1.35;
    }

    timing(x, N);
    
    free(x);
    return 0;
}

void timing(const real *x, const int N)
{
    real sum = 0;

    for (int i = 0; i < NUM_REPEATS; ++i) {
        hipEvent_t start, end;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&end));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(x, N);

        CHECK(hipEventRecord(end));
        CHECK(hipEventSynchronize(end));

        float elpased_time;
        CHECK(hipEventElapsedTime(&elpased_time, start, end));
        printf("Time = %g ms.\n", elpased_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(end));
    }

    printf("sum = %f.\n", sum);
}

real reduce(const real *x, const int N)
{
    real sum = 0.0;
    for (int i = 0; i < N; ++i) {
        sum += x[i];
    }
    return sum;
}